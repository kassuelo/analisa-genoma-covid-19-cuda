#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define TAM 30800

//-----aloca memória global - na RAM e na GPU
__managed__ char vetor1[TAM];
__managed__ char vetor2[TAM];
__managed__ int qtd_iguais[10][10];

//------Kernel que sera executado na GPU
__global__ void compara_genoma(int *qtde)
{
	//a cada comparação de 2 arquivos serão iniciadas 30800 threads
	//cada thread ira comparar um caracter de cada arquivo
	// 30800 caracteres x 45 comparações = 1386000 threads iniciadas até o final das 45 comparações entre os 10 arquivos
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ((vetor1[idx] != NULL) || (vetor2[idx] != NULL))
	{
		if (vetor1[idx] == vetor2[idx])
		{
			atomicAdd(qtde, 1);
		}
	}
	vetor1[idx] = NULL;
	vetor2[idx] = NULL;
}

void mostra_iguais()
{
    int maior_qtd_iguais=0;
    int arq1;
    int arq2;
    printf("\n MATRIZ IGUAIS\n");
    printf("\n arq0 | arq1 | arq2 | arq3 | arq4 | arq5 | arq6 | arq7 | arq8 | arq9 |\n\n");
    for (int i = 0; i < 10; i++)
    {
        for (int j = 0; j < 10; j++)
        {
            if(maior_qtd_iguais<qtd_iguais[i][j]){
                maior_qtd_iguais= qtd_iguais[i][j];
                arq1=i;
                arq2=j;
            }
            printf(" %5d|", qtd_iguais[i][j]);
        }
        printf("\n");
    }
    printf("\nMaior quantidade de iguais = %d, entre os genomas %d e %d\n\n",maior_qtd_iguais,arq1,arq2);
}

int main(int argc, char *argv[0])
{
	//-----cria uma var do tipo ponteiro
	int *num;
	//-----aloca memória na RAM e na GPU
	hipMallocManaged(&num, 4);

	//-----inicializa endereço do ponteiro com 0
	*num = 0;
	srand(time(NULL));

	for (int j = 0; j < 9; j++)
	{
		for (int k = j + 1; k < 10; k++)
		{

			// Initializa vetor na CPU
			// envia dinamicamente o nome do arquivo de deve ser aberto
			char g1[14] = "genomas/";
			g1[8] = j + '0';
			strcat(g1, ".txt");
			char g2[14] = "genomas/";
			g2[8] = k + '0';
			strcat(g2, ".txt");

			char c; //guarda o caracter lido
			//abre o primeiro
			FILE *file1;
			file1 = fopen(g1, "r");
			int i = 0;
			while ((c = getc(file1)) != EOF)
			{
				vetor1[i] = c;
				i++;
			}
			fclose(file1);

			//abre o segundo arquivo
			FILE *file2;
			file2 = fopen(g2, "r");
			i = 0;
			while ((c = getc(file2)) != EOF)
			{
				vetor2[i] = c;
				i++;
			}
			fclose(file2);

			//-----programa principal exibe o vetor inicial
			printf("*COMPARANDO O ARQUIVO %d e o ARQUIVO %d", j, k);

			//-----executa a função compara_genoma na GPU com 30800 threads
			compara_genoma<<<700, 44>>>(num);

			//-----cria uma barreira - espera todas as threads finalizarem
			hipDeviceSynchronize();

			qtd_iguais[j][k] = *num;
			*num = 0;
			printf("\n\n");
		}
	}
	mostra_iguais();
}
